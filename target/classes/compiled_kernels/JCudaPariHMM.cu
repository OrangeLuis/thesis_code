#include "hip/hip_runtime.h"
extern "C"
__global__ double subComputation(byte [] haplotypeBases, byte [] readBases,
                                 byte [] readQuals, byte [] insertionGOP,
                                 byte [] deletionGOP, byte [] overall GCP,
                                 int, hapStartIndex, boolean recacheReadValues,
                                 int nextHapStartIndex) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] + b[i];
    }
}