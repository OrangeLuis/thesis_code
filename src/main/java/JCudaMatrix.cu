#include "hip/hip_runtime.h"
extern "C"
__global__ void subComputation(int numElements, float [] *row, float [] *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for(int j = 0; j < numElements; j++)
    {
        sum[j] += row[i][j];
    }
}